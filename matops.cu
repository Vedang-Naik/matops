#include "hip/hip_runtime.h"
#include <stdio.h>
extern "C" {
#include "matops.h"
}

void printMatrix(Matrix A) {
	for (int i = 0; i < A.rows; i++) {
		for (int j = 0; j < A.cols; j++) {
			printf("%.2f ", *(A.elements + i*A.cols + j));
		}
		printf("\n");
	}
	printf("\n");
}

Matrix initMatrix(int rows, int cols, float* elements) {
	if (rows <= 0 || cols <= 0) {
		printf("Please use positive values for rows and columns.\n");
		exit(0);
	}

	Matrix temp;
	temp.rows = rows;
	temp.cols = cols;
	temp.elements = (float *)malloc(rows * cols * sizeof(float));
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			*(temp.elements + i*cols + j) = *(elements + i*cols + j);
		}
	}

	return temp;
}

//================================================================================//

__global__ void pam(Matrix A_d, Matrix B_d, Matrix C_d) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	*(C_d.elements + i*C_d.cols + j) = *(A_d.elements + i*A_d.cols + j) + *(B_d.elements + i*B_d.cols + j);
}

Matrix pAddMatrix(Matrix A, Matrix B) {
	if (A.rows != B.rows || A.cols != B.cols) {
		printf("Your input matrices do not have equal dimensions.\n");
		exit(0);
	}

	int numRows = A.rows;
	int numCols = A.cols;
	int sizeOfMatrix = numRows * numCols * sizeof(float);

	Matrix A_d;
	A_d.rows = numRows;
	A_d.cols = numCols;
	hipMalloc(&A_d.elements, sizeOfMatrix);
	hipMemcpy(A_d.elements, A.elements, sizeOfMatrix, hipMemcpyHostToDevice);

	Matrix B_d;
	B_d.rows = numRows;
	B_d.cols = numCols;
	hipMalloc(&B_d.elements, sizeOfMatrix);
	hipMemcpy(B_d.elements, B.elements, sizeOfMatrix, hipMemcpyHostToDevice);

	Matrix C_d;
	C_d.rows = numRows;
	C_d.cols = numCols;
	hipMalloc(&C_d.elements, sizeOfMatrix);
	pam<<<1, dim3(numRows, numCols)>>>(A_d, B_d, C_d);

	Matrix C;
	C.rows = numRows;
	C.cols = numCols;
	C.elements = (float *)malloc(sizeOfMatrix);
	hipMemcpy(C.elements, C_d.elements, sizeOfMatrix, hipMemcpyDeviceToHost);

	hipFree(A_d.elements);
	hipFree(B_d.elements);
	hipFree(C_d.elements);

	return C;
}

//================================================================================//

__global__ void psm(Matrix A_d, Matrix B_d, Matrix C_d) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	*(C_d.elements + i*C_d.cols + j) = *(A_d.elements + i*A_d.cols + j) - *(B_d.elements + i*B_d.cols + j);
}

Matrix pSubMatrix(Matrix A, Matrix B) {
	if (A.rows != B.rows || A.cols != B.cols) {
		printf("Your input matrices do not have equal dimensions.\n");
		exit(0);
	}

	int numRows = A.rows;
	int numCols = A.cols;
	int sizeOfMatrix = numRows * numCols * sizeof(float);

	Matrix A_d;
	A_d.rows = numRows;
	A_d.cols = numCols;
	hipMalloc(&A_d.elements, sizeOfMatrix);
	hipMemcpy(A_d.elements, A.elements, sizeOfMatrix, hipMemcpyHostToDevice);

	Matrix B_d;
	B_d.rows = numRows;
	B_d.cols = numCols;
	hipMalloc(&B_d.elements, sizeOfMatrix);
	hipMemcpy(B_d.elements, B.elements, sizeOfMatrix, hipMemcpyHostToDevice);

	Matrix C_d;
	C_d.rows = numRows;
	C_d.cols = numCols;
	hipMalloc(&C_d.elements, sizeOfMatrix);
	pam<<<1, dim3(numRows, numCols)>>>(A_d, B_d, C_d);

	Matrix C;
	C.rows = numRows;
	C.cols = numCols;
	C.elements = (float *)malloc(sizeOfMatrix);
	hipMemcpy(C.elements, C_d.elements, sizeOfMatrix, hipMemcpyDeviceToHost);

	hipFree(A_d.elements);
	hipFree(B_d.elements);
	hipFree(C_d.elements);

	return C;
}

//================================================================================//

float* getRow(Matrix A, int i) {
	if (i > A.rows || i < 1) {
		printf("Your row number is out of bounds.\n");
		exit(0);
	}

	float* reqRowPtr = &*(A.elements + (i-1)*A.cols);
	return reqRowPtr;
}

float* getCol(Matrix A, int j) {
	if (j > A.cols || j < 1) {
		printf("Your column number is out of bounds.\n");
		exit(0);
	}

	float* reqColPtr = (float* )malloc(A.cols * sizeof(float));
	for (int i = 0; i < A.rows; i++) {
		*(reqColPtr + i) = *(A.elements + i*A.cols + (j-1));
	}
	return reqColPtr;
}

float getElm(Matrix A, int i, int j) {
	if (i > A.rows || i < 1 || j > A.cols || j < 1) {
		printf("Your row or column number is out of bounds.\n");
		exit(0);
	}

	return *(A.elements + (i-1)*A.cols + (j-1));
}